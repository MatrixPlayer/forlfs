#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <iostream>

#include "caffe/layers/relu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data_output = bottom[0]->cpu_data();
  std::cout << "in Relu" << std::endl;
  std::cout << bottom_data_output[0] << std::endl;
  std::cout << bottom_data_output[1] << std::endl;
  std::cout << "out Relu" << std::endl;
  //FILE* fcout = fopen("/mnt/data1/zdb/work/reid/ssd_caffe/caffe/log_debug.txt", "a+");
  //fprintf(fcout, "in RELU \n");
  //for(int id = 0; id < 4; id++) {
  //  fprintf(fcout, "%f\n", bottom_data_output[id]);
  //}
  //fprintf(fcout, "out RELU \n");
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;

  //std::cout  << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS << std::endl;
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope);
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
